#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include "mink.cuh"


template <typename scalar_t>
__global__ void knn_aggregate_forward_cuda_kernel(
    const torch::PackedTensorAccessor32<scalar_t, 3, torch::RestrictPtrTraits> q,
    const torch::PackedTensorAccessor32<scalar_t, 3, torch::RestrictPtrTraits> p,
    const torch::PackedTensorAccessor32<scalar_t, 3, torch::RestrictPtrTraits> f,
    const torch::PackedTensorAccessor32<scalar_t, 3, torch::RestrictPtrTraits> sigma,
    torch::PackedTensorAccessor32<scalar_t, 3, torch::RestrictPtrTraits> f_out,
    torch::PackedTensorAccessor32<scalar_t, 3, torch::RestrictPtrTraits> w_out,
    scalar_t* min_dists,
    int64_t* min_idxs,
    const int K,
    const int F
) {
    const int b = blockIdx.x * blockDim.x + threadIdx.x;
    const int n_q = blockIdx.y * blockDim.y + threadIdx.y;

    // skip extraneous compute
    if (b >= q.size(0) || n_q >= q.size(1)) return;
    int offset = b * q.size(1) * K + n_q * K;

    // create a data structure for holding the min-k
    MinK<scalar_t, int64_t> mink(min_dists + offset, min_idxs + offset, K);

    const int P = p.size(1);
    // iterate through all surface points
    for (int p_idx = 0; p_idx < P; ++ p_idx) {
        // iterate through all dimensions (assume q and p is 3 dimension)
        scalar_t sq_dist = 0;
        for (int d = 0; d < 3; ++d) {
            const scalar_t diff = q[b][n_q][d] - p[b][p_idx][d];
            sq_dist += diff * diff;
        }
        mink.add(sq_dist, p_idx);
    }
    // sort it so everyone is happy, O(K^2) but K is small anyway
    mink.sort();

    // iterate through the min-k elements to acquire the aggregated features
    for (int k = 0; k < mink.size(); ++k) {
        int64_t k_idx = mink.val(k);
        const scalar_t w = __expf(-mink.key(k) * sigma[b][k_idx][0]);

        // go through the feature
        for (int n_f = 0; n_f < F; ++n_f){
            f_out[b][n_q][n_f] += f[b][k_idx][n_f] * w;
        }
        w_out[b][n_q][0] += w;
    }
}


template <typename scalar_t>
__global__ void knn_aggregate_aniso_forward_cuda_kernel(
    const torch::PackedTensorAccessor32<scalar_t, 3, torch::RestrictPtrTraits> q,
    const torch::PackedTensorAccessor32<scalar_t, 3, torch::RestrictPtrTraits> p,
    const torch::PackedTensorAccessor32<scalar_t, 3, torch::RestrictPtrTraits> f,
    const torch::PackedTensorAccessor32<scalar_t, 3, torch::RestrictPtrTraits> sigma,
    const torch::PackedTensorAccessor32<scalar_t, 4, torch::RestrictPtrTraits> R,
    torch::PackedTensorAccessor32<scalar_t, 3, torch::RestrictPtrTraits> f_out,
    torch::PackedTensorAccessor32<scalar_t, 3, torch::RestrictPtrTraits> w_out,
    scalar_t* min_dists,
    int64_t* min_idxs,
    const int K,
    const int F
) {
    const int b = blockIdx.x * blockDim.x + threadIdx.x;
    const int n_q = blockIdx.y * blockDim.y + threadIdx.y;

    // skip extraneous compute
    if (b >= q.size(0) || n_q >= q.size(1)) return;
    // f_out[b][n_q] += torch::matmul(R[b][n_q], f[b][n_q]);
    int offset = b * q.size(1) * K + n_q * K;

    // create a data structure for holding the min-k
    MinK<scalar_t, int64_t> mink(min_dists + offset, min_idxs + offset, K);

    const int P = p.size(1);
    // iterate through all surface points
    for (int p_idx = 0; p_idx < P; ++p_idx) {

        scalar_t x = q[b][n_q][0] - p[b][p_idx][0];
        scalar_t y = q[b][n_q][1] - p[b][p_idx][1];
        scalar_t z = q[b][n_q][2] - p[b][p_idx][2];
        scalar_t j = R[b][p_idx][0][0] * x + R[b][p_idx][0][1] * y + R[b][p_idx][0][2] * z;
        scalar_t k = R[b][p_idx][1][0] * x + R[b][p_idx][1][1] * y + R[b][p_idx][1][2] * z;
        scalar_t l = R[b][p_idx][2][0] * x + R[b][p_idx][2][1] * y + R[b][p_idx][2][2] * z;
        // compute "scaled" square distance ... not sure if this is the best way to do it.
        scalar_t sq_dist = j * j * sigma[b][p_idx][0] + k * k * sigma[b][p_idx][1] + l * l * sigma[b][p_idx][2];

        mink.add(sq_dist, p_idx);
    }
    // sort it so everyone is happy, O(K^2) but K is small anyway
    mink.sort();

    // iterate through the min-k elements to acquire the aggregated features
    for (int k = 0; k < mink.size(); ++k) {
        int64_t k_idx = mink.val(k);
        const scalar_t w = __expf(-mink.key(k));

        // go through the feature
        for (int n_f = 0; n_f < F; ++n_f){
            f_out[b][n_q][n_f] += f[b][k_idx][n_f] * w;
        }
        w_out[b][n_q][k] = w;
    }
}


template <typename scalar_t>
__global__ void knn_lookup_aggregate_aniso_forward_cuda_kernel(
    const torch::PackedTensorAccessor32<scalar_t, 3, torch::RestrictPtrTraits> q,
    const torch::PackedTensorAccessor32<scalar_t, 3, torch::RestrictPtrTraits> p,
    const torch::PackedTensorAccessor32<scalar_t, 3, torch::RestrictPtrTraits> f,
    const torch::PackedTensorAccessor32<scalar_t, 3, torch::RestrictPtrTraits> sigma,
    const torch::PackedTensorAccessor32<scalar_t, 4, torch::RestrictPtrTraits> R,
    const torch::PackedTensorAccessor32<int64_t, 2, torch::RestrictPtrTraits> knn_table,
    torch::PackedTensorAccessor32<scalar_t, 3, torch::RestrictPtrTraits> f_out,
    torch::PackedTensorAccessor32<scalar_t, 3, torch::RestrictPtrTraits> w_out,
    scalar_t* min_dists,
    int64_t* min_idxs,
    const int K,
    const int F
) {
    const int b = blockIdx.x * blockDim.x + threadIdx.x;
    const int n_q = blockIdx.y * blockDim.y + threadIdx.y;

    // skip extraneous compute
    if (b >= q.size(0) || n_q >= q.size(1)) return;
    // f_out[b][n_q] += torch::matmul(R[b][n_q], f[b][n_q]);
    int offset = b * q.size(1) * K + n_q * K;


    const int P = p.size(1);
    scalar_t min_dist = 1e10;
    int64_t min_idx = 0;
    // iterate through all surface points
    for (int p_idx = 0; p_idx < P; ++ p_idx) {

        scalar_t x = q[b][n_q][0] - p[b][p_idx][0];
        scalar_t y = q[b][n_q][1] - p[b][p_idx][1];
        scalar_t z = q[b][n_q][2] - p[b][p_idx][2];
        /*
        // now these are only computed for the tabluated points, should save a bunch of computes?
        scalar_t j = R[b][p_idx][0][0] * x + R[b][p_idx][0][1] * y + R[b][p_idx][0][2] * z;
        scalar_t k = R[b][p_idx][1][0] * x + R[b][p_idx][1][1] * y + R[b][p_idx][1][2] * z;
        scalar_t l = R[b][p_idx][2][0] * x + R[b][p_idx][2][1] * y + R[b][p_idx][2][2] * z;
        // compute "scaled" square distance ... not sure if this is the best way to do it.
        scalar_t sq_dist = j * j * sigma[b][p_idx][0] + k * k * sigma[b][p_idx][1] + l * l * sigma[b][p_idx][2];
        */
        scalar_t sq_dist = x * x + y * y + z * z;
        if (sq_dist < min_dist) {
            min_dist = sq_dist;
            min_idx = p_idx;
        }

    }
    // sort it so everyone is happy, O(K^2) but K is small anyway

    // iterate through the min-k elements to acquire the aggregated features
    for (int n_k = 0; n_k < K; ++n_k) {
        int64_t k_idx = knn_table[min_idx][n_k];
        scalar_t x = q[b][n_q][0] - p[b][k_idx][0];
        scalar_t y = q[b][n_q][1] - p[b][k_idx][1];
        scalar_t z = q[b][n_q][2] - p[b][k_idx][2];
        scalar_t j = R[b][k_idx][0][0] * x + R[b][k_idx][0][1] * y + R[b][k_idx][0][2] * z;
        scalar_t k = R[b][k_idx][1][0] * x + R[b][k_idx][1][1] * y + R[b][k_idx][1][2] * z;
        scalar_t l = R[b][k_idx][2][0] * x + R[b][k_idx][2][1] * y + R[b][k_idx][2][2] * z;
        scalar_t sq_dist = j * j * sigma[b][k_idx][0] + k * k * sigma[b][k_idx][1] + l * l * sigma[b][k_idx][2];
        const scalar_t w = __expf(-sq_dist);

        // go through the feature
        for (int n_f = 0; n_f < F; ++n_f){
            f_out[b][n_q][n_f] += f[b][k_idx][n_f] * w;
        }
        w_out[b][n_q][n_k] = w;
        min_idxs[offset + n_k] = k_idx;
        min_dists[offset + n_k] = sq_dist;
    }
}


template <typename scalar_t>
__global__ void knn_precomputed_aggregate_aniso_forward_cuda_kernel(
    const torch::PackedTensorAccessor32<scalar_t, 3, torch::RestrictPtrTraits> q,
    const torch::PackedTensorAccessor32<scalar_t, 3, torch::RestrictPtrTraits> p,
    const torch::PackedTensorAccessor32<scalar_t, 3, torch::RestrictPtrTraits> f,
    const torch::PackedTensorAccessor32<scalar_t, 3, torch::RestrictPtrTraits> sigma,
    const torch::PackedTensorAccessor32<scalar_t, 4, torch::RestrictPtrTraits> R,
    const torch::PackedTensorAccessor32<int64_t, 3, torch::RestrictPtrTraits> knn_idxs,
    torch::PackedTensorAccessor32<scalar_t, 3, torch::RestrictPtrTraits> f_out,
    torch::PackedTensorAccessor32<scalar_t, 3, torch::RestrictPtrTraits> w_out,
    scalar_t* min_dists,
    const int K,
    const int F
) {
    const int b = blockIdx.x * blockDim.x + threadIdx.x;
    const int n_q = blockIdx.y * blockDim.y + threadIdx.y;

    // skip extraneous compute
    if (b >= q.size(0) || n_q >= q.size(1)) return;
    // f_out[b][n_q] += torch::matmul(R[b][n_q], f[b][n_q]);
    int offset = b * q.size(1) * K + n_q * K;
    const int P = p.size(1);

    // sort it so everyone is happy, O(K^2) but K is small anyway

    // iterate through the min-k elements to acquire the aggregated features
    for (int n_k = 0; n_k < K; ++n_k) {
        int64_t k_idx = knn_idxs[b][n_q][n_k];
        scalar_t x = q[b][n_q][0] - p[b][k_idx][0];
        scalar_t y = q[b][n_q][1] - p[b][k_idx][1];
        scalar_t z = q[b][n_q][2] - p[b][k_idx][2];
        scalar_t j = R[b][k_idx][0][0] * x + R[b][k_idx][0][1] * y + R[b][k_idx][0][2] * z;
        scalar_t k = R[b][k_idx][1][0] * x + R[b][k_idx][1][1] * y + R[b][k_idx][1][2] * z;
        scalar_t l = R[b][k_idx][2][0] * x + R[b][k_idx][2][1] * y + R[b][k_idx][2][2] * z;
        scalar_t sq_dist = j * j * sigma[b][k_idx][0] + k * k * sigma[b][k_idx][1] + l * l * sigma[b][k_idx][2];
        const scalar_t w = __expf(-sq_dist);

        // go through the feature
        for (int n_f = 0; n_f < F; ++n_f){
            f_out[b][n_q][n_f] += f[b][k_idx][n_f] * w;
        }
        w_out[b][n_q][n_k] = w;
        min_dists[offset + n_k] = sq_dist;
    }
}


std::vector<torch::Tensor> knn_aggregate_forward_cuda(
    const torch::Tensor q,
    const torch::Tensor p,
    const torch::Tensor f,
    const torch::Tensor sigma,
    const int K
) {
    /*
        q: (B, N_query, 3)
        p: (B, N_surface_pts, 3)
        f: (B, N_surface_pts, C)
        sigma: (B, N_surface_pts, 1) // TODO: how to aniso?
     */
    const int B = q.size(0);
    const int Q = q.size(1);
    const int F = f.size(2);
    // TODO: revisit to see if other configs run faster 
    const dim3 threads(16, 16); // use a total of 256 threads per-block
    const dim3 blocks((B + threads.x - 1) / threads.x, (Q + threads.y - 1) / threads.y);

    // f.options: set data type and device
    // to specify a particular dtype torch::zeros({N, F}, torch::dtype(torch::kInt32).device(feats.device));
    torch::Tensor f_out = torch::zeros({B, Q, F}, f.options());
    torch::Tensor w_out = torch::zeros({B, Q, 1}, f.options());

    // empty goes faster
    torch::Tensor min_dists = torch::empty({B, Q, K}, q.options());
    torch::Tensor min_idxs = torch::empty({B, Q, K}, torch::dtype(torch::kInt64).device(f.device()));

    // float32 or float64
    // AT_DISPATCH_FLOATING_TYPES_HALF -- float16
    // [&]: captures all variables in the function scope by reference --> [...] {} is lambda function
    // RestrictPtrTratis: pointers will not overlap
    AT_DISPATCH_FLOATING_TYPES(f.scalar_type(), "knn_aggregate_forward_cuda_kernel", ([&] {
        knn_aggregate_forward_cuda_kernel<scalar_t><<<blocks, threads>>>(
            q.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>(),
            p.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>(),
            f.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>(),
            sigma.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>(),
            f_out.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>(),
            w_out.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>(),
            min_dists.data_ptr<scalar_t>(),
            min_idxs.data_ptr<int64_t>(),
            K,
            f.size(2)
        );
    }));
    // TODO: whatelse do we need for backward?
    //return {f_out, f_cache_out, w_out, min_dists, min_idxs};
    return {f_out, w_out, min_dists, min_idxs};
}


std::vector<torch::Tensor> knn_aggregate_aniso_forward_cuda(
    const torch::Tensor q,
    const torch::Tensor p,
    const torch::Tensor f,
    const torch::Tensor sigma,
    const torch::Tensor R,
    const int K
) {
    /*
        q: (B, N_query, 3)
        p: (B, N_surface_pts, 3)
        f: (B, N_surface_pts, C)
        sigma: (B, N_surface_pts, 3, 3) 
        R: (B, N_surface_pts, 3, 3)
     */
    const int B = q.size(0);
    const int Q = q.size(1);
    const int F = f.size(2);
    // TODO: revisit to see if other configs run faster 
    const dim3 threads(16, 16); // use a total of 256 threads per-block
    const dim3 blocks((B + threads.x - 1) / threads.x, (Q + threads.y - 1) / threads.y);

    // f.options: set data type and device
    // to specify a particular dtype torch::zeros({N, F}, torch::dtype(torch::kInt32).device(feats.device));
    torch::Tensor f_out = torch::zeros({B, Q, F}, f.options());

    // empty goes faster
    torch::Tensor w_out = torch::empty({B, Q, K}, f.options());
    torch::Tensor min_dists = torch::empty({B, Q, K}, q.options());
    torch::Tensor min_idxs = torch::empty({B, Q, K}, torch::dtype(torch::kInt64).device(f.device()));

    // float32 or float64
    // AT_DISPATCH_FLOATING_TYPES_HALF -- float16
    // [&]: captures all variables in the function scope by reference --> [...] {} is lambda function
    // RestrictPtrTratis: pointers will not overlap
    AT_DISPATCH_FLOATING_TYPES(f.scalar_type(), "knn_aggregate_aniso_forward_cuda_kernel", ([&] {
        knn_aggregate_aniso_forward_cuda_kernel<scalar_t><<<blocks, threads>>>(
            q.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>(),
            p.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>(),
            f.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>(),
            sigma.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>(),
            R.packed_accessor32<scalar_t, 4, torch::RestrictPtrTraits>(),
            f_out.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>(),
            w_out.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>(),
            min_dists.data_ptr<scalar_t>(),
            min_idxs.data_ptr<int64_t>(),
            K,
            f.size(2)
        );
    }));
    // TODO: whatelse do we need for backward?
    return {f_out, w_out, min_dists, min_idxs};
}


std::vector<torch::Tensor> knn_lookup_aggregate_aniso_forward_cuda(
    const torch::Tensor q,
    const torch::Tensor p,
    const torch::Tensor f,
    const torch::Tensor sigma,
    const torch::Tensor R,
    const torch::Tensor knn_table,
    const int K
) {
    /*
        q: (B, N_query, 3)
        p: (B, N_surface_pts, 3)
        f: (B, N_surface_pts, C)
        sigma: (B, N_surface_pts, 3, 3) 
        R: (B, N_surface_pts, 3, 3)
     */
    const int B = q.size(0);
    const int Q = q.size(1);
    const int F = f.size(2);
    // TODO: revisit to see if other configs run faster 
    const dim3 threads(16, 16); // use a total of 256 threads per-block
    const dim3 blocks((B + threads.x - 1) / threads.x, (Q + threads.y - 1) / threads.y);

    // f.options: set data type and device
    // to specify a particular dtype torch::zeros({N, F}, torch::dtype(torch::kInt32).device(feats.device));
    torch::Tensor f_out = torch::zeros({B, Q, F}, f.options());

    // empty goes faster
    torch::Tensor w_out = torch::empty({B, Q, K}, f.options());
    torch::Tensor min_dists = torch::empty({B, Q, K}, q.options());
    torch::Tensor min_idxs = torch::empty({B, Q, K}, torch::dtype(torch::kInt64).device(f.device()));

    // float32 or float64
    // AT_DISPATCH_FLOATING_TYPES_HALF -- float16
    // [&]: captures all variables in the function scope by reference --> [...] {} is lambda function
    // RestrictPtrTratis: pointers will not overlap
    AT_DISPATCH_FLOATING_TYPES(f.scalar_type(), "knn_lookup_aggregate_aniso_forward_cuda_kernel", ([&] {
        knn_lookup_aggregate_aniso_forward_cuda_kernel<scalar_t><<<blocks, threads>>>(
            q.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>(),
            p.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>(),
            f.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>(),
            sigma.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>(),
            R.packed_accessor32<scalar_t, 4, torch::RestrictPtrTraits>(),
            knn_table.packed_accessor32<int64_t, 2, torch::RestrictPtrTraits>(),
            f_out.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>(),
            w_out.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>(),
            min_dists.data_ptr<scalar_t>(),
            min_idxs.data_ptr<int64_t>(),
            K,
            f.size(2)
        );
    }));
    // TODO: whatelse do we need for backward?
    return {f_out, w_out, min_dists, min_idxs};
}


std::vector<torch::Tensor> knn_precomputed_aggregate_aniso_forward_cuda(
    const torch::Tensor q,
    const torch::Tensor p,
    const torch::Tensor f,
    const torch::Tensor sigma,
    const torch::Tensor R,
    const torch::Tensor knn_idxs,
    const int K
) {
    /*
        q: (B, N_query, 3)
        p: (B, N_surface_pts, 3)
        f: (B, N_surface_pts, C)
        sigma: (B, N_surface_pts, 3, 3) 
        R: (B, N_surface_pts, 3, 3)
     */
    const int B = q.size(0);
    const int Q = q.size(1);
    const int F = f.size(2);
    // TODO: revisit to see if other configs run faster 
    const dim3 threads(16, 16); // use a total of 256 threads per-block
    const dim3 blocks((B + threads.x - 1) / threads.x, (Q + threads.y - 1) / threads.y);

    // f.options: set data type and device
    // to specify a particular dtype torch::zeros({N, F}, torch::dtype(torch::kInt32).device(feats.device));
    torch::Tensor f_out = torch::zeros({B, Q, F}, f.options());

    // empty goes faster
    torch::Tensor w_out = torch::empty({B, Q, K}, f.options());
    torch::Tensor min_dists = torch::empty({B, Q, K}, q.options());

    // float32 or float64
    // AT_DISPATCH_FLOATING_TYPES_HALF -- float16
    // [&]: captures all variables in the function scope by reference --> [...] {} is lambda function
    // RestrictPtrTratis: pointers will not overlap
    AT_DISPATCH_FLOATING_TYPES(f.scalar_type(), "knn_precomputed_aggregate_aniso_forward_cuda_kernel", ([&] {
        knn_precomputed_aggregate_aniso_forward_cuda_kernel<scalar_t><<<blocks, threads>>>(
            q.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>(),
            p.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>(),
            f.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>(),
            sigma.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>(),
            R.packed_accessor32<scalar_t, 4, torch::RestrictPtrTraits>(),
            knn_idxs.packed_accessor32<int64_t, 3, torch::RestrictPtrTraits>(),
            f_out.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>(),
            w_out.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>(),
            min_dists.data_ptr<scalar_t>(),
            K,
            f.size(2)
        );
    }));
    // TODO: whatelse do we need for backward?
    return {f_out, w_out, min_dists};
}
