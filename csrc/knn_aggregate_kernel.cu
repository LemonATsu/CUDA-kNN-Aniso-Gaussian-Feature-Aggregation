#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include "mink.cuh"
#include <iostream>

template <typename scalar_t>
__global__ void knn_aggregate_forward_cuda_kernel(
    const torch::PackedTensorAccessor32<scalar_t, 3, torch::RestrictPtrTraits> q,
    const torch::PackedTensorAccessor32<scalar_t, 3, torch::RestrictPtrTraits> p,
    const torch::PackedTensorAccessor32<scalar_t, 3, torch::RestrictPtrTraits> f,
    const torch::PackedTensorAccessor32<scalar_t, 3, torch::RestrictPtrTraits> sigma,
    torch::PackedTensorAccessor32<scalar_t, 3, torch::RestrictPtrTraits> f_out,
    torch::PackedTensorAccessor32<scalar_t, 3, torch::RestrictPtrTraits> w_out,
    scalar_t* min_dists,
    int64_t* min_idxs,
    const int K,
    const int F
) {
    const int b = blockIdx.x * blockDim.x + threadIdx.x;
    const int n_q = blockIdx.y * blockDim.y + threadIdx.y;

    // skip extraneous compute
    if (b >= q.size(0) || n_q >= q.size(1)) return;
    int offset = b * q.size(1) * K + n_q * K;

    // create a data structure for holding the min-k
    MinK<scalar_t, int64_t> mink(min_dists + offset, min_idxs + offset, K);

    const int P = p.size(1);
    // iterate through all surface points
    for (int p_idx = 0; p_idx < P; ++ p_idx) {
        // iterate through all dimensions (assume q and p is 3 dimension)
        scalar_t sq_dist = 0;
        for (int d = 0; d < 3; ++d) {
            const scalar_t diff = q[b][n_q][d] - p[b][p_idx][d];
            sq_dist += diff * diff;
        }
        mink.add(sq_dist, p_idx);
    }
    // sort it so everyone is happy, O(K^2) but K is small anyway
    mink.sort();

    // iterate through the min-k elements to acquire the aggregated features
    for (int k = 0; k < mink.size(); ++k) {
        int64_t k_idx = mink.val(k);
        const scalar_t w = __expf(-mink.key(k) * sigma[b][k_idx][0]);

        // go through the feature
        for (int n_f = 0; n_f < F; ++n_f){
            f_out[b][n_q][n_f] += f[b][k_idx][n_f] * w;
        }
        w_out[b][n_q][0] += w;
    }
}


template <typename scalar_t>
__global__ void knn_aggregate_aniso_forward_cuda_kernel(
    const torch::PackedTensorAccessor32<scalar_t, 3, torch::RestrictPtrTraits> q,
    const torch::PackedTensorAccessor32<scalar_t, 3, torch::RestrictPtrTraits> p,
    const torch::PackedTensorAccessor32<scalar_t, 3, torch::RestrictPtrTraits> f,
    const torch::PackedTensorAccessor32<scalar_t, 3, torch::RestrictPtrTraits> sigma,
    const torch::PackedTensorAccessor32<scalar_t, 4, torch::RestrictPtrTraits> R,
    torch::PackedTensorAccessor32<scalar_t, 3, torch::RestrictPtrTraits> f_out,
    torch::PackedTensorAccessor32<scalar_t, 3, torch::RestrictPtrTraits> w_out,
    scalar_t* min_dists,
    int64_t* min_idxs,
    const int K,
    const int F
) {
    const int b = blockIdx.x * blockDim.x + threadIdx.x;
    const int n_q = blockIdx.y * blockDim.y + threadIdx.y;

    // skip extraneous compute
    if (b >= q.size(0) || n_q >= q.size(1)) return;
    // f_out[b][n_q] += torch::matmul(R[b][n_q], f[b][n_q]);
    int offset = b * q.size(1) * K + n_q * K;

    // create a data structure for holding the min-k
    MinK<scalar_t, int64_t> mink(min_dists + offset, min_idxs + offset, K);

    const int P = p.size(1);
    // iterate through all surface points
    for (int p_idx = 0; p_idx < P; ++ p_idx) {

        scalar_t x = q[b][n_q][0] - p[b][p_idx][0];
        scalar_t y = q[b][n_q][1] - p[b][p_idx][1];
        scalar_t z = q[b][n_q][2] - p[b][p_idx][2];
        scalar_t j = R[b][p_idx][0][0] * x + R[b][p_idx][0][1] * y + R[b][p_idx][0][2] * z;
        scalar_t k = R[b][p_idx][1][0] * x + R[b][p_idx][1][1] * y + R[b][p_idx][1][2] * z;
        scalar_t l = R[b][p_idx][2][0] * x + R[b][p_idx][2][1] * y + R[b][p_idx][2][2] * z;
        // compute "scaled" square distance ... not sure if this is the best way to do it.
        scalar_t sq_dist = j * j * sigma[b][p_idx][0] + k * k * sigma[b][p_idx][1] + l * l * sigma[b][p_idx][2];

        mink.add(sq_dist, p_idx);
    }
    // sort it so everyone is happy, O(K^2) but K is small anyway
    mink.sort();

    // iterate through the min-k elements to acquire the aggregated features
    for (int k = 0; k < mink.size(); ++k) {
        int64_t k_idx = mink.val(k);
        const scalar_t w = __expf(-mink.key(k));

        // go through the feature
        for (int n_f = 0; n_f < F; ++n_f){
            f_out[b][n_q][n_f] += f[b][k_idx][n_f] * w;
        }
        w_out[b][n_q][k] = w;
    }
}


template <typename scalar_t>
__global__ void knn_aggregate_aniso_backward_cuda_kernel(
    const torch::PackedTensorAccessor32<scalar_t, 3, torch::RestrictPtrTraits> grad_f_out,
    const torch::PackedTensorAccessor32<scalar_t, 3, torch::RestrictPtrTraits> grad_w_out,
    const torch::PackedTensorAccessor32<scalar_t, 3, torch::RestrictPtrTraits> q,
    const torch::PackedTensorAccessor32<scalar_t, 3, torch::RestrictPtrTraits> p,
    const torch::PackedTensorAccessor32<scalar_t, 3, torch::RestrictPtrTraits> f,
    const torch::PackedTensorAccessor32<scalar_t, 3, torch::RestrictPtrTraits> sigma,
    const torch::PackedTensorAccessor32<scalar_t, 4, torch::RestrictPtrTraits> R,
    const torch::PackedTensorAccessor32<scalar_t, 3, torch::RestrictPtrTraits> w_out,
    const torch::PackedTensorAccessor64<int64_t, 3, torch::RestrictPtrTraits> k_idxs,
    torch::PackedTensorAccessor32<scalar_t, 3, torch::RestrictPtrTraits> dLdq,
    torch::PackedTensorAccessor32<scalar_t, 3, torch::RestrictPtrTraits> dLdp,
    torch::PackedTensorAccessor32<scalar_t, 3, torch::RestrictPtrTraits> dLdf,
    torch::PackedTensorAccessor32<scalar_t, 3, torch::RestrictPtrTraits> dLdsigma,
    const int K,
    const int F
) {
    // each thread handle one query point, and does atomic operation
    const int b = blockIdx.x * blockDim.x + threadIdx.x;
    const int n_q = blockIdx.y * blockDim.y + threadIdx.y;

    // skip extraneous compute
    if (b >= q.size(0) || n_q >= q.size(1)) return;

    for (int n_k = 0; n_k < K; ++n_k) {
        int64_t k_idx = k_idxs[b][n_q][n_k]; // get the n-th ponit index

        // prepare the quantities we need
        scalar_t x = q[b][n_q][0] - p[b][k_idx][0];
        scalar_t y = q[b][n_q][1] - p[b][k_idx][1];
        scalar_t z = q[b][n_q][2] - p[b][k_idx][2];

        // apply rotation to the difference
        scalar_t j = R[b][k_idx][0][0] * x + R[b][k_idx][0][1] * y + R[b][k_idx][0][2] * z;
        scalar_t k = R[b][k_idx][1][0] * x + R[b][k_idx][1][1] * y + R[b][k_idx][1][2] * z;
        scalar_t l = R[b][k_idx][2][0] * x + R[b][k_idx][2][1] * y + R[b][k_idx][2][2] * z;
        scalar_t w = w_out[b][n_q][n_k];

        // compyte dwdp
        scalar_t dwdpx = -2 * w * (
            -j * R[b][k_idx][0][0] * sigma[b][k_idx][0] +
            -k * R[b][k_idx][1][0] * sigma[b][k_idx][1] +
            -l * R[b][k_idx][2][0] * sigma[b][k_idx][2]
        );

        scalar_t dwdpy = -2 * w * (
            -j * R[b][k_idx][0][1] * sigma[b][k_idx][0] +
            -k * R[b][k_idx][1][1] * sigma[b][k_idx][1] +
            -l * R[b][k_idx][2][1] * sigma[b][k_idx][2]
        );

        scalar_t dwdpz = -2 * w * (
            -j * R[b][k_idx][0][2] * sigma[b][k_idx][0] +
            -k * R[b][k_idx][1][2] * sigma[b][k_idx][1] +
            -l * R[b][k_idx][2][2] * sigma[b][k_idx][2]
        );

        scalar_t j_sq = j * j;
        scalar_t k_sq = k * k;
        scalar_t l_sq = l * l;

        scalar_t dwdsigmax = -w * j_sq;
        scalar_t dwdsigmay = -w * k_sq;
        scalar_t dwdsigmaz = -w * l_sq;

        /////////////////////////////
        //     dLdsigma and dLdf   //
        /////////////////////////////
        for (int n_f = 0; n_f < F; ++n_f) {
            // dwdsigma = -w * d/dsigma(exp(...)) = -w * j^2
            // dL1dsigma = (dL1dfo * dfodw + dLdw) * dwdsigma
            scalar_t grad_fo = grad_f_out[b][n_q][n_f];
            scalar_t dL1dfo_dfodw = grad_fo * f[b][k_idx][n_f]; 

            // use atomicAdd to avoid race condition
            // dLdq and dLdp is differ by a negative sign
            atomicAdd(&dLdq[b][n_q][0], dL1dfo_dfodw * -dwdpx);
            atomicAdd(&dLdq[b][n_q][1], dL1dfo_dfodw * -dwdpy);
            atomicAdd(&dLdq[b][n_q][2], dL1dfo_dfodw * -dwdpz);

            atomicAdd(&dLdp[b][k_idx][0], dL1dfo_dfodw * dwdpx);
            atomicAdd(&dLdp[b][k_idx][1], dL1dfo_dfodw * dwdpy);
            atomicAdd(&dLdp[b][k_idx][2], dL1dfo_dfodw * dwdpz);

            atomicAdd(&dLdf[b][k_idx][n_f], grad_fo * w);

            atomicAdd(&dLdsigma[b][k_idx][0], dL1dfo_dfodw * dwdsigmax);
            atomicAdd(&dLdsigma[b][k_idx][1], dL1dfo_dfodw * dwdsigmay);
            atomicAdd(&dLdsigma[b][k_idx][2], dL1dfo_dfodw * dwdsigmaz);

        }
        // add gradient from dL2dw, i.e., the case where we use w_out somewhere
        scalar_t dL2dw = grad_w_out[b][n_q][n_k]; 
        atomicAdd(&dLdq[b][n_q][0], dL2dw * -dwdpx);
        atomicAdd(&dLdq[b][n_q][1], dL2dw * -dwdpy);
        atomicAdd(&dLdq[b][n_q][2], dL2dw * -dwdpz);

        atomicAdd(&dLdp[b][k_idx][0], dL2dw * dwdpx);
        atomicAdd(&dLdp[b][k_idx][1], dL2dw * dwdpy);
        atomicAdd(&dLdp[b][k_idx][2], dL2dw * dwdpz);

        atomicAdd(&dLdsigma[b][k_idx][0], dL2dw * dwdsigmax);
        atomicAdd(&dLdsigma[b][k_idx][1], dL2dw * dwdsigmay);
        atomicAdd(&dLdsigma[b][k_idx][2], dL2dw * dwdsigmaz);

    }
}


template <typename scalar_t>
__global__ void knn_aggregate_aniso_backward_2nd_cuda_kernel(
    const torch::PackedTensorAccessor32<scalar_t, 3, torch::RestrictPtrTraits> grad_grad_q,
    const torch::PackedTensorAccessor32<scalar_t, 3, torch::RestrictPtrTraits> grad_f_out,
    const torch::PackedTensorAccessor32<scalar_t, 3, torch::RestrictPtrTraits> grad_w_out,
    const torch::PackedTensorAccessor32<scalar_t, 3, torch::RestrictPtrTraits> q,
    const torch::PackedTensorAccessor32<scalar_t, 3, torch::RestrictPtrTraits> p,
    const torch::PackedTensorAccessor32<scalar_t, 3, torch::RestrictPtrTraits> f,
    const torch::PackedTensorAccessor32<scalar_t, 3, torch::RestrictPtrTraits> sigma,
    const torch::PackedTensorAccessor32<scalar_t, 4, torch::RestrictPtrTraits> R,
    const torch::PackedTensorAccessor32<scalar_t, 3, torch::RestrictPtrTraits> w_out,
    const torch::PackedTensorAccessor64<int64_t, 3, torch::RestrictPtrTraits> k_idxs,
    torch::PackedTensorAccessor32<scalar_t, 3, torch::RestrictPtrTraits> dLdp,
    torch::PackedTensorAccessor32<scalar_t, 3, torch::RestrictPtrTraits> dLdf,
    torch::PackedTensorAccessor32<scalar_t, 3, torch::RestrictPtrTraits> dLdsigma,
    const int K,
    const int F
) {
    // each thread handle one query point, and does atomic operation
    const int b = blockIdx.x * blockDim.x + threadIdx.x;
    const int n_q = blockIdx.y * blockDim.y + threadIdx.y;

    // skip extraneous compute
    if (b >= q.size(0) || n_q >= q.size(1)) return;
    scalar_t dLddqx = grad_grad_q[b][n_q][0];
    scalar_t dLddqy = grad_grad_q[b][n_q][1];
    scalar_t dLddqz = grad_grad_q[b][n_q][2];

    for (int n_k = 0; n_k < K; ++n_k) {
        int64_t k_idx = k_idxs[b][n_q][n_k]; // get the n-th ponit index

        // prepare the quantities we need
        scalar_t x = q[b][n_q][0] - p[b][k_idx][0];
        scalar_t y = q[b][n_q][1] - p[b][k_idx][1];
        scalar_t z = q[b][n_q][2] - p[b][k_idx][2];

        // apply rotation to the difference
        scalar_t j = R[b][k_idx][0][0] * x + R[b][k_idx][0][1] * y + R[b][k_idx][0][2] * z;
        scalar_t k = R[b][k_idx][1][0] * x + R[b][k_idx][1][1] * y + R[b][k_idx][1][2] * z;
        scalar_t l = R[b][k_idx][2][0] * x + R[b][k_idx][2][1] * y + R[b][k_idx][2][2] * z;

        // these quantities will be used again
        scalar_t sigma_x = sigma[b][k_idx][0];
        scalar_t sigma_y = sigma[b][k_idx][1];
        scalar_t sigma_z = sigma[b][k_idx][2];
        scalar_t Ra = R[b][k_idx][0][0];
        scalar_t Rb = R[b][k_idx][0][1];
        scalar_t Rc = R[b][k_idx][0][2];
        scalar_t Rd = R[b][k_idx][1][0];
        scalar_t Re = R[b][k_idx][1][1];
        scalar_t Rf = R[b][k_idx][1][2];
        scalar_t Rg = R[b][k_idx][2][0];
        scalar_t Rh = R[b][k_idx][2][1];
        scalar_t Ri = R[b][k_idx][2][2];

        scalar_t sigmax_a = sigma_x * Ra;
        scalar_t sigmax_b = sigma_x * Rb;
        scalar_t sigmax_c = sigma_x * Rc;
        scalar_t sigmay_d = sigma_y * Rd;
        scalar_t sigmay_e = sigma_y * Re;
        scalar_t sigmay_f = sigma_y * Rf;
        scalar_t sigmaz_g = sigma_z * Rg;
        scalar_t sigmaz_h = sigma_z * Rh;
        scalar_t sigmaz_i = sigma_z * Ri;


        scalar_t dotx = (j * sigmax_a + k * sigmay_d + l * sigmaz_g);
        scalar_t doty = (j * sigmax_b + k * sigmay_e + l * sigmaz_h);
        scalar_t dotz = (j * sigmax_c + k * sigmay_f + l * sigmaz_i);

        // these are the same as in 1st order
        // TODO: can be replaced by for loop, but do we want to do that?

        scalar_t w = w_out[b][n_q][n_k];
        scalar_t d_dwdqx_dqx = (2 * dotx * dotx - (sigmax_a * Ra + sigmay_d * Rd + sigmaz_g * Rg));
        scalar_t d_dwdqx_dqy = (2 * doty * dotx - (sigmax_a * Rb + sigmay_d * Re + sigmaz_g * Rh));
        scalar_t d_dwdqx_dqz = (2 * dotz * dotx - (sigmax_a * Rc + sigmay_d * Rf + sigmaz_g * Ri));

        scalar_t d_dwdqy_dqx = (2 * dotx * doty - (sigmax_b * Ra + sigmay_e * Rd + sigmaz_h * Rg));
        scalar_t d_dwdqy_dqy = (2 * doty * doty - (sigmax_b * Rb + sigmay_e * Re + sigmaz_h * Rh));
        scalar_t d_dwdqy_dqz = (2 * dotz * doty - (sigmax_b * Rc + sigmay_e * Rf + sigmaz_h * Ri));

        scalar_t d_dwdqz_dqx = (2 * dotx * dotz - (sigmax_c * Ra + sigmay_f * Rd + sigmaz_i * Rg));
        scalar_t d_dwdqz_dqy = (2 * doty * dotz - (sigmax_c * Rb + sigmay_f * Re + sigmaz_i * Rh));
        scalar_t d_dwdqz_dqz = (2 * dotz * dotz - (sigmax_c * Rc + sigmay_f * Rf + sigmaz_i * Ri));

        scalar_t d_dwdqx_dsigmax = 2 * w * j * (j * dotx - Ra);
        scalar_t d_dwdqy_dsigmax = 2 * w * j * (j * doty - Rb);
        scalar_t d_dwdqz_dsigmax = 2 * w * j * (j * dotz - Rc);

        scalar_t d_dwdqx_dsigmay = 2 * w * k * (k * dotx - Rd);
        scalar_t d_dwdqy_dsigmay = 2 * w * k * (k * doty - Re);
        scalar_t d_dwdqz_dsigmay = 2 * w * k * (k * dotz - Rf);

        scalar_t d_dwdqx_dsigmaz = 2 * w * l * (l * dotx - Rg);
        scalar_t d_dwdqy_dsigmaz = 2 * w * l * (l * doty - Rh);
        scalar_t d_dwdqz_dsigmaz = 2 * w * l * (l * dotz - Ri);

        /////////////////////////////
        //     dLdsigma and dLdf   //
        /////////////////////////////
        for (int n_f = 0; n_f < F; ++n_f) {
            // same as before, but need to multiply by dLddqx/y/z
            scalar_t grad_fo = grad_f_out[b][n_q][n_f];
            scalar_t dL1dfo_dfodw = grad_fo * f[b][k_idx][n_f]; 

            // use atomicAdd to avoid race condition
            //atomicAdd(&dLdp[b][k_idx][0], dLddqx * dL1dfo_dfodw * (d_dwdqx_dqx + d_dwdqy_dqx + d_dwdqz_dqx));
            //atomicAdd(&dLdp[b][k_idx][1], dLddqy * dL1dfo_dfodw * (d_dwdqx_dqy + d_dwdqy_dqy + d_dwdqz_dqy));
            //atomicAdd(&dLdp[b][k_idx][2], dLddqz * dL1dfo_dfodw * (d_dwdqx_dqz + d_dwdqy_dqz + d_dwdqz_dqz));

            atomicAdd(&dLdp[b][k_idx][0], dL1dfo_dfodw * -2 * w * (dLddqx * d_dwdqx_dqx + dLddqy * d_dwdqy_dqx + dLddqz * d_dwdqz_dqx));
            atomicAdd(&dLdp[b][k_idx][1], dL1dfo_dfodw * -2 * w * (dLddqx * d_dwdqx_dqy + dLddqy * d_dwdqy_dqy + dLddqz * d_dwdqz_dqy));
            atomicAdd(&dLdp[b][k_idx][2], dL1dfo_dfodw * -2 * w * (dLddqx * d_dwdqx_dqz + dLddqy * d_dwdqy_dqz + dLddqz * d_dwdqz_dqz));

            atomicAdd(&dLdsigma[b][k_idx][0], dL1dfo_dfodw * (dLddqx * d_dwdqx_dsigmax + dLddqy * d_dwdqy_dsigmax + dLddqz * d_dwdqz_dsigmax));
            atomicAdd(&dLdsigma[b][k_idx][1], dL1dfo_dfodw * (dLddqx * d_dwdqx_dsigmay + dLddqy * d_dwdqy_dsigmay + dLddqz * d_dwdqz_dsigmay));
            atomicAdd(&dLdsigma[b][k_idx][2], dL1dfo_dfodw * (dLddqx * d_dwdqx_dsigmaz + dLddqy * d_dwdqy_dsigmaz + dLddqz * d_dwdqz_dsigmaz));
        }
    }
}


std::vector<torch::Tensor> knn_aggregate_forward_cuda(
    const torch::Tensor q,
    const torch::Tensor p,
    const torch::Tensor f,
    const torch::Tensor sigma,
    const int K
) {
    /*
        q: (B, N_query, 3)
        p: (B, N_surface_pts, 3)
        f: (B, N_surface_pts, C)
        sigma: (B, N_surface_pts, 1) // TODO: how to aniso?
     */
    const int B = q.size(0);
    const int Q = q.size(1);
    const int F = f.size(2);
    // TODO: revisit to see if other configs run faster 
    const dim3 threads(16, 16); // use a total of 256 threads per-block
    const dim3 blocks((B + threads.x - 1) / threads.x, (Q + threads.y - 1) / threads.y);

    // f.options: set data type and device
    // to specify a particular dtype torch::zeros({N, F}, torch::dtype(torch::kInt32).device(feats.device));
    torch::Tensor f_out = torch::zeros({B, Q, F}, f.options());
    torch::Tensor w_out = torch::zeros({B, Q, 1}, f.options());

    // empty goes faster
    torch::Tensor min_dists = torch::empty({B, Q, K}, q.options());
    torch::Tensor min_idxs = torch::empty({B, Q, K}, torch::dtype(torch::kInt64).device(f.device()));

    // float32 or float64
    // AT_DISPATCH_FLOATING_TYPES_HALF -- float16
    // [&]: captures all variables in the function scope by reference --> [...] {} is lambda function
    // RestrictPtrTratis: pointers will not overlap
    AT_DISPATCH_FLOATING_TYPES(f.scalar_type(), "knn_aggregate_forward_cuda_kernel", ([&] {
        knn_aggregate_forward_cuda_kernel<scalar_t><<<blocks, threads>>>(
            q.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>(),
            p.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>(),
            f.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>(),
            sigma.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>(),
            f_out.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>(),
            w_out.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>(),
            min_dists.data_ptr<scalar_t>(),
            min_idxs.data_ptr<int64_t>(),
            K,
            f.size(2)
        );
    }));
    // TODO: whatelse do we need for backward?
    //return {f_out, f_cache_out, w_out, min_dists, min_idxs};
    return {f_out, w_out, min_dists, min_idxs};
}


std::vector<torch::Tensor> knn_aggregate_aniso_forward_cuda(
    const torch::Tensor q,
    const torch::Tensor p,
    const torch::Tensor f,
    const torch::Tensor sigma,
    const torch::Tensor R,
    const int K
) {
    /*
        q: (B, N_query, 3)
        p: (B, N_surface_pts, 3)
        f: (B, N_surface_pts, C)
        sigma: (B, N_surface_pts, 3, 3) 
        R: (B, N_surface_pts, 3, 3)
     */
    const int B = q.size(0);
    const int Q = q.size(1);
    const int F = f.size(2);
    // TODO: revisit to see if other configs run faster 
    const dim3 threads(16, 16); // use a total of 256 threads per-block
    const dim3 blocks((B + threads.x - 1) / threads.x, (Q + threads.y - 1) / threads.y);

    // f.options: set data type and device
    // to specify a particular dtype torch::zeros({N, F}, torch::dtype(torch::kInt32).device(feats.device));
    torch::Tensor f_out = torch::zeros({B, Q, F}, f.options());

    // empty goes faster
    torch::Tensor w_out = torch::empty({B, Q, K}, f.options());
    torch::Tensor min_dists = torch::empty({B, Q, K}, q.options());
    torch::Tensor min_idxs = torch::empty({B, Q, K}, torch::dtype(torch::kInt64).device(f.device()));

    // float32 or float64
    // AT_DISPATCH_FLOATING_TYPES_HALF -- float16
    // [&]: captures all variables in the function scope by reference --> [...] {} is lambda function
    // RestrictPtrTratis: pointers will not overlap
    AT_DISPATCH_FLOATING_TYPES(f.scalar_type(), "knn_aggregate_aniso_forward_cuda_kernel", ([&] {
        knn_aggregate_aniso_forward_cuda_kernel<scalar_t><<<blocks, threads>>>(
            q.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>(),
            p.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>(),
            f.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>(),
            sigma.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>(),
            R.packed_accessor32<scalar_t, 4, torch::RestrictPtrTraits>(),
            f_out.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>(),
            w_out.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>(),
            min_dists.data_ptr<scalar_t>(),
            min_idxs.data_ptr<int64_t>(),
            K,
            f.size(2)
        );
    }));
    // TODO: whatelse do we need for backward?
    return {f_out, w_out, min_dists, min_idxs};
}


std::vector<torch::Tensor> knn_aggregate_aniso_backward_cuda(
    const torch::Tensor grad_f_out,
    const torch::Tensor grad_w_out,
    const torch::Tensor q,
    const torch::Tensor p,
    const torch::Tensor f,
    const torch::Tensor sigma,
    const torch::Tensor R,
    const torch::Tensor w_out,
    const torch::Tensor k_idxs
) {
    /*
        grad_f_out: (B, N_query, C) -- gradient w.r.t aggregated features
        grad_w_out: (B, N_query, K) -- gradient w.r.t kNN weights
        q: (B, N_query, 3)
        p: (B, N_surface_pts, 3)
        f: (B, N_surface_pts, C)
        sigma: (B, N_surface_pts, 3, 3) 
        R: (B, N_surface_pts, 3, 3)
     */
    const int B = q.size(0);
    const int Q = q.size(1);
    const int P = p.size(1);
    const int F = f.size(2);
    // TODO: revisit to see if other configs run faster 
    const dim3 threads(16, 16); // use a total of 256 threads per-block
    const dim3 blocks((B + threads.x - 1) / threads.x, (Q + threads.y - 1) / threads.y);

    // we need grad_p, grad_f, and grad_sigma
    torch::Tensor dLdq = torch::zeros({B, Q, 3}, q.options());
    torch::Tensor dLdp = torch::zeros({B, P, 3}, p.options());
    torch::Tensor dLdf = torch::zeros({B, P, F}, f.options());
    torch::Tensor dLdsigma = torch::zeros({B, P, 3}, sigma.options());

    int K = k_idxs.size(2);
    AT_DISPATCH_FLOATING_TYPES(f.scalar_type(), "knn_aggregate_aniso_backward_cuda_kernel", ([&] {
        knn_aggregate_aniso_backward_cuda_kernel<scalar_t><<<blocks, threads>>>(
            grad_f_out.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>(),
            grad_w_out.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>(),
            q.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>(),
            p.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>(),
            f.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>(),
            sigma.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>(),
            R.packed_accessor32<scalar_t, 4, torch::RestrictPtrTraits>(),
            w_out.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>(),
            k_idxs.packed_accessor64<int64_t, 3, torch::RestrictPtrTraits>(),
            dLdq.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>(),
            dLdp.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>(),
            dLdf.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>(),
            dLdsigma.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>(),
            K,
            F
        );
    }));
    // TODO: whatelse do we need for backward?
    return {dLdq, dLdp, dLdf, dLdsigma};
}


std::vector<torch::Tensor> knn_aggregate_aniso_backward_2nd_cuda(
    const torch::Tensor grad_grad_q,
    const torch::Tensor grad_f_out,
    const torch::Tensor grad_w_out,
    const torch::Tensor q,
    const torch::Tensor p,
    const torch::Tensor f,
    const torch::Tensor sigma,
    const torch::Tensor R,
    const torch::Tensor w_out,
    const torch::Tensor k_idxs
) {
    /*
        grad_grad_q: (B, N_query, 3)
        grad_grad_p: (B, N_surface_pts, 3)
        grad_f_out: (B, N_query, C) -- gradient w.r.t aggregated features
        grad_w_out: (B, N_query, K) -- gradient w.r.t kNN weights
        q: (B, N_query, 3)
        p: (B, N_surface_pts, 3)
        f: (B, N_surface_pts, C)
        sigma: (B, N_surface_pts, 3, 3) 
        R: (B, N_surface_pts, 3, 3)
     */
    const int B = q.size(0);
    const int Q = q.size(1);
    const int P = p.size(1);
    const int F = f.size(2);
    // TODO: revisit to see if other configs run faster 
    const dim3 threads(16, 16); // use a total of 256 threads per-block
    const dim3 blocks((B + threads.x - 1) / threads.x, (Q + threads.y - 1) / threads.y);

    // we need grad_p, grad_f, and grad_sigma again
    torch::Tensor dLdp = torch::zeros({B, P, 3}, p.options());
    torch::Tensor dLdf = torch::zeros({B, P, F}, f.options());
    torch::Tensor dLdsigma = torch::zeros({B, P, 3}, sigma.options());

    int K = k_idxs.size(2);
    AT_DISPATCH_FLOATING_TYPES(f.scalar_type(), "knn_aggregate_aniso_backward_2nd_cuda_kernel", ([&] {
        knn_aggregate_aniso_backward_2nd_cuda_kernel<scalar_t><<<blocks, threads>>>(
            grad_grad_q.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>(),
            grad_f_out.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>(),
            grad_w_out.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>(),
            q.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>(),
            p.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>(),
            f.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>(),
            sigma.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>(),
            R.packed_accessor32<scalar_t, 4, torch::RestrictPtrTraits>(),
            w_out.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>(),
            k_idxs.packed_accessor64<int64_t, 3, torch::RestrictPtrTraits>(),
            dLdp.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>(),
            dLdf.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>(),
            dLdsigma.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>(),
            K,
            F
        );
    }));
    return {dLdp, dLdf, dLdsigma};
}